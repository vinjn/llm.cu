#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float* x, float* y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void)
{
    FILE* fp = fopen("../data/gpt-2.safetensors", "r");
    if (fp == NULL) {
        printf("Error opening file!\n");
        exit(1);
    }

    // Read the first 10 bytes of the file
    char buffer[10];
    size_t bytesRead = fread(buffer, 1, 10, fp);
    if (bytesRead < 10) {
        printf("Error reading file!\n");
        exit(1);
    }


    int N = 1 << 20; // 1M elements

    // Allocate Unified Memory �C accessible from CPU or GPU
    float* x, * y;
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add << <numBlocks, blockSize >> > (N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}